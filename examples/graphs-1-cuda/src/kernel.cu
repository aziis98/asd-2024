
#include <hip/hip_runtime.h>
extern "C" __global__ void compute_forces(
    float *positions_x, float *positions_y,
    float *forces_x, float *forces_y,
    int *neighbors, float *distances,
    int num_nodes, int max_neighbors)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_nodes)
    {
        float pos_x = positions_x[idx];
        float pos_y = positions_y[idx];
        float force_x = 0.0f;
        float force_y = 0.0f;

        for (int i = 0; i < max_neighbors; i++)
        {
            int other_idx = neighbors[idx * max_neighbors + i];
            if (other_idx == -1)
                break; // No more neighbors

            float other_pos_x = positions_x[other_idx];
            float other_pos_y = positions_y[other_idx];
            float distance = distances[idx * max_neighbors + i];

            float delta_x = other_pos_x - pos_x;
            float delta_y = other_pos_y - pos_y;
            float dist = delta_x * delta_x + delta_y * delta_y;
            float correction = dist - (distance * distance);

            if (distance > 0.0f && dist > 1e-6f)
            {
                float scale = 0.01f * atanf(correction) / sqrtf(dist);
                force_x += delta_x * scale;
                force_y += delta_y * scale;
            }

            if (dist > 1e-6f)
            {
                float repel_scale = 0.01f / max(dist, 1.0f);
                force_x -= delta_x * repel_scale;
                force_y -= delta_y * repel_scale;
            }
        }

        forces_x[idx] = force_x;
        forces_y[idx] = force_y;
    }
}
